
#include <hip/hip_runtime.h>
#include "iostream"
#include <fstream>
#include <string>

#define BLOCK_SIZE  16          // submatrix size
#define N           1024        // matrix size is N*N

__global__ void matMult ( float * a, float * b, int n, float * c )
{
    int   bx  = blockIdx.x;     // block index
    int   by  = blockIdx.y;
    int   tx  = threadIdx.x;        // thread index
    int   ty  = threadIdx.y;
    float sum = 0.0f;           // computed subelement
    int   ia  = n * BLOCK_SIZE * by + n * ty;   // a [i][0]
    int   ib  = BLOCK_SIZE * bx + tx;

    // Multiply the two matrices together;
    for ( int k = 0; k < n; k++ )
        sum += a [ia + k] * b [ib + k*n];

    // Write the block sub-matrix to global memory;
    // each thread writes one element
    int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;

    c [ic + n * ty + tx] = sum;
}

void write_data_to_file(std::string messge, float * a){
    std::ofstream myfile;
    myfile.open ("test_data.txt", std::fstream::app);
    myfile << messge + "\n";
    for ( int i = 0; i < N; i++ )
        for ( int j = 0; j < N; j++ )
        {
            int	k = N*i + j;
            myfile << a [k];
        }
    myfile << "\n";
    myfile.close();
}

void clear_file(){
    std::ofstream ofs;
    ofs.open("test_data.txt", std::ofstream::out | std::ofstream::trunc);
    ofs.close();
}

int main ( int argc, char *  argv [] )
{
    clear_file();
    int numBytes = N * N * sizeof ( float );

    // allocate host memory
    float * a = new float [N*N];
    float * b = new float [N*N];
    float * c = new float [N*N];

    for ( int i = 0; i < N; i++ )
        for ( int j = 0; j < N; j++ )
        {
            int	k = N*i + j;

            a [k] = 0.0f;
            b [k] = 1.0f;
        }
    write_data_to_file("First matrix", a);
    write_data_to_file("Second matrix", b);

    // allocate device memory
    float * adev = NULL;
    float * bdev = NULL;
    float * cdev = NULL;

    hipMalloc ( (void**)&adev, numBytes );
    hipMalloc ( (void**)&bdev, numBytes );
    hipMalloc ( (void**)&cdev, numBytes );

    // set kernel launch configuration
    dim3 threads ( BLOCK_SIZE, BLOCK_SIZE );
    dim3 blocks  ( N / threads.x, N / threads.y);

    // create cuda event handles
    hipEvent_t start, stop;
    float gpuTime = 0.0f;

    hipEventCreate ( &start );
    hipEventCreate ( &stop );

    // asynchronously issue work to the GPU (all to stream 0)
    hipEventRecord ( start, 0 );
    hipMemcpy      ( adev, a, numBytes, hipMemcpyHostToDevice );
    hipMemcpy      ( bdev, b, numBytes, hipMemcpyHostToDevice );

    matMult<<<blocks, threads>>> ( adev, bdev, N, cdev );

    hipMemcpy      ( c, cdev, numBytes, hipMemcpyDeviceToHost );
    hipEventRecord ( stop, 0 );

    hipEventSynchronize ( stop );
    hipEventElapsedTime ( &gpuTime, start, stop );

    // print the cpu and gpu times
    printf("time spent executing by the GPU: %.2f millseconds, results were saved to test_data.txt\n", gpuTime );
    write_data_to_file("Result matrix", c);

    // release resources
    hipEventDestroy ( start );
    hipEventDestroy ( stop  );
    hipFree         ( adev  );
    hipFree         ( bdev  );
    hipFree         ( cdev  );

    delete a;
    delete b;
    delete c;

    return 0;
}